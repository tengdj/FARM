#include "hip/hip_runtime.h"
#include "cuda_util.h"
#include "../include/Ideal.h"
#include "mygpu.h"

void cuda_create_buffer(query_context *gctx){
    hipSetDevice(1);

	unsigned long long size = BUFFER_SIZE;
    log("CPU momory:");

    gctx->h_info = (Idealinfo*)new char[size / 4ULL];
    log("\t%.2f MB\tideal info buffer",1.0*size/1024/1024/4);
     
	gctx->h_status = new uint8_t[size / 4ULL];
    log("\t%.2f MB\tstatus buffer",1.0*size/1024/1024/4);

    gctx->h_offset = (uint16_t*)new char[size / 2ULL];
    log("\t%.2f MB\toffset buffer",1.0*size/1024/1024/2);

    gctx->h_edge_sequences = (EdgeSeq *)new char[size];
    log("\t%.2f MB\tedge sequences buffer",1.0*size/1024/1024);

    gctx->h_vertices = (Point *)new char[4ULL * size];
    log("\t%.2f MB\tvertices buffer",4.0*size/1024/1024);

	log("GPU memory:");
    CUDA_SAFE_CALL(hipMalloc((void **) &gctx->d_info, size/4ULL));
	log("\t%.2f MB\tideal info buffer",1.0*size/1024/1024/4);

    CUDA_SAFE_CALL(hipMalloc((void **) &gctx->d_status, size/4ULL));
	log("\t%.2f MB\tstatus buffer",1.0*size/1024/1024/4);

    CUDA_SAFE_CALL(hipMalloc((void **) &gctx->d_offset, size/2ULL));
	log("\t%.2f MB\toffset buffer",1.0*size/1024/1024/2);
    
    CUDA_SAFE_CALL(hipMalloc((void **) &gctx->d_edge_sequences, size));
	log("\t%.2f MB\tedge sequences buffer",1.0*size/1024/1024);

    CUDA_SAFE_CALL(hipMalloc((void **) &gctx->d_vertices, size * 4ULL));
	log("\t%.2f MB\tvertices buffer",4.0*size/1024/1024);

}

void preprocess_for_gpu(query_context *gctx){
    hipSetDevice(1);
    // compact data
    uint iidx = 0, sidx = 0, oidx = 0, eidx = 0, vidx = 0;
    for(auto &tp : gctx->ideal_pairs){
        Ideal *source = tp.first;
        int dimx = source->get_dimx(), dimy = source->get_dimy();
        if(source->idealoffset == nullptr){
            source->idealoffset = new IdealOffset{};

            uint info_size = gctx->ideal_pairs.size();
            Idealinfo idealinfo{source->getMBB(), dimx, dimy, source->get_step_x(), source->get_step_y()};
            memcpy(gctx->h_info+iidx, &idealinfo, sizeof(Idealinfo));
            source->idealoffset->info_start = iidx;
            iidx ++;
            source->idealoffset->info_end = iidx;

            uint status_size = (dimx+1)*(dimy+1) / 4 + 1;
            memcpy(gctx->h_status+sidx, source->get_status(), status_size);
            source->idealoffset->status_start = sidx;
            sidx += status_size;
            source->idealoffset->status_end = sidx;

            uint offset_size = (dimx+1)*(dimy+1) + 1;
            memcpy(gctx->h_offset+oidx, source->get_offset(), offset_size * sizeof(uint16_t));
            source->idealoffset->offset_start = oidx;
            oidx += offset_size;
            source->idealoffset->offset_end = oidx;

            uint edge_sequences_size = source->get_len_edge_sequences();
            memcpy(gctx->h_edge_sequences+eidx, source->get_edge_sequence(), edge_sequences_size * sizeof(EdgeSeq));
            source->idealoffset->edge_sequences_start = eidx;
            eidx += edge_sequences_size;
            source->idealoffset->edge_sequences_end = eidx;

            uint vertices_size = source->get_num_vertices();
            memcpy(gctx->h_vertices+vidx, source->get_boundary()->p, vertices_size * sizeof(Point));
            source->idealoffset->vertices_start = vidx;
            vidx += vertices_size;
            source->idealoffset->vertices_end = vidx;
        }

        Ideal *target = tp.second;   
	    dimx = target->get_dimx(), dimy = target->get_dimy();
        if(target->idealoffset == nullptr){
            target->idealoffset = new IdealOffset{};

            uint info_size = gctx->ideal_pairs.size();
            Idealinfo idealinfo{target->getMBB(), dimx, dimy, target->get_step_x(), target->get_step_y()};
            memcpy(gctx->h_info+iidx, &idealinfo, sizeof(Idealinfo));
            target->idealoffset->info_start = iidx;
            iidx ++;
            target->idealoffset->info_end = iidx;

            uint status_size = (dimx+1)*(dimy+1) / 4 + 1;
            assert((status_size+sidx) < 1U * BUFFER_SIZE);
            memcpy(gctx->h_status+sidx, target->get_status(), status_size);
            target->idealoffset->status_start = sidx;
            sidx += status_size;
            target->idealoffset->status_end = sidx;

            uint offset_size = (dimx+1)*(dimy+1) + 1;
            memcpy(gctx->h_offset+oidx, target->get_offset(), offset_size * sizeof(uint16_t));
            target->idealoffset->offset_start = oidx;
            oidx += offset_size;
            target->idealoffset->offset_end = oidx;

            uint edge_sequences_size = target->get_len_edge_sequences();
            assert((edge_sequences_size+eidx)*sizeof(EdgeSeq) < 1U * BUFFER_SIZE);
            memcpy(gctx->h_edge_sequences+eidx, target->get_edge_sequence(), edge_sequences_size * sizeof(EdgeSeq));
            target->idealoffset->edge_sequences_start = eidx;
            eidx += edge_sequences_size;
            target->idealoffset->edge_sequences_end = eidx;

            uint vertices_size = target->get_num_vertices();
            assert((vertices_size+vidx)*sizeof(Point) < (4ULL * BUFFER_SIZE));
            memcpy(gctx->h_vertices+vidx, target->get_boundary()->p, vertices_size * sizeof(Point));
            target->idealoffset->vertices_start = vidx;
            vidx += vertices_size;
            target->idealoffset->vertices_end = vidx;
        }
    }

    CUDA_SAFE_CALL(hipMemcpy(gctx->d_info, gctx->h_info, BUFFER_SIZE / 4UL * sizeof(uint8_t), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(gctx->d_status, gctx->h_status, BUFFER_SIZE / 4UL * sizeof(uint8_t), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(gctx->d_offset, gctx->h_offset, BUFFER_SIZE / 2UL * sizeof(uint8_t), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(gctx->d_edge_sequences, gctx->h_edge_sequences, BUFFER_SIZE * sizeof(uint8_t), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(gctx->d_vertices, gctx->h_vertices, 4UL * BUFFER_SIZE * sizeof(uint8_t), hipMemcpyHostToDevice));
}