#include "hip/hip_runtime.h"
#include "cuda_util.h"
#include "../include/Ideal.h"
#include "mygpu.h"

void cuda_create_buffer(query_context *gctx){
    hipSetDevice(1);

	unsigned long long size = BUFFER_SIZE;
    log("CPU momory:");

    gctx->h_info = (Idealinfo*)new char[size / 4ULL];
    log("\t%.2f MB\tideal info buffer",1.0*size/1024/1024/4);
     
	gctx->h_status = new uint8_t[size / 4ULL];
    log("\t%.2f MB\tstatus buffer",1.0*size/1024/1024/4);

    gctx->h_offset = (uint16_t*)new char[size / 2ULL];
    log("\t%.2f MB\toffset buffer",1.0*size/1024/1024/2);

    gctx->h_edge_sequences = (EdgeSeq *)new char[size];
    log("\t%.2f MB\tedge sequences buffer",1.0*size/1024/1024);

    gctx->h_vertices = (Point *)new char[4ULL * size];
    log("\t%.2f MB\tvertices buffer",4.0*size/1024/1024);

    gctx->h_gridline_offset = (uint16_t *)new char[size / 4ULL];
    log("\t%.2f MB\tgrid line offset buffer",1.0*size/1024/1024/4);

    gctx->h_gridline_nodes = (double *)new char[size / 4ULL];
    log("\t%.2f MB\tgrid line nodes buffer",1.0*size/1024/1024/4);

	log("GPU memory:");
    CUDA_SAFE_CALL(hipMalloc((void **) &gctx->d_info, size/4ULL));
	log("\t%.2f MB\tideal info buffer",1.0*size/1024/1024/4);

    CUDA_SAFE_CALL(hipMalloc((void **) &gctx->d_status, size/4ULL));
	log("\t%.2f MB\tstatus buffer",1.0*size/1024/1024/4);

    CUDA_SAFE_CALL(hipMalloc((void **) &gctx->d_offset, size/2ULL));
	log("\t%.2f MB\toffset buffer",1.0*size/1024/1024/2);
    
    CUDA_SAFE_CALL(hipMalloc((void **) &gctx->d_edge_sequences, size));
	log("\t%.2f MB\tedge sequences buffer",1.0*size/1024/1024);

    CUDA_SAFE_CALL(hipMalloc((void **) &gctx->d_vertices, size * 4ULL));
	log("\t%.2f MB\tvertices buffer",4.0*size/1024/1024);

    CUDA_SAFE_CALL(hipMalloc((void **) &gctx->d_gridline_offset, size/4ULL));    
    log("\t%.2f MB\tgrid line offset buffer",1.0*size/1024/1024/4);

    CUDA_SAFE_CALL(hipMalloc((void **) &gctx->d_gridline_nodes, size/4ULL));    
    log("\t%.2f MB\tgrid line nodes buffer",1.0*size/1024/1024/4);

}

void preprocess_for_gpu(query_context *gctx){
    hipSetDevice(1);
    bool flag1 = false, flag2 = false;
    // compact data
    uint iidx = 0, sidx = 0, oidx = 0, eidx = 0, vidx = 0, goidx = 0, gnidx = 0;
    for(auto &tp : gctx->point_polygon_pairs){
        flag1 = true; 
        Ideal *source = tp.second;
        int dimx = source->get_dimx(), dimy = source->get_dimy();
        if(source->idealoffset == nullptr){
            source->idealoffset = new IdealOffset{};

            uint info_size = gctx->polygon_pairs.size();
            Idealinfo idealinfo{source->getMBB(), dimx, dimy, source->get_step_x(), source->get_step_y()};
            memcpy(gctx->h_info+iidx, &idealinfo, sizeof(Idealinfo));
            source->idealoffset->info_start = iidx;
            iidx ++;
            source->idealoffset->info_end = iidx;

            uint status_size = (dimx+1)*(dimy+1) / 4 + 1;
            memcpy(gctx->h_status+sidx, source->get_status(), status_size);
            source->idealoffset->status_start = sidx;
            sidx += status_size;
            source->idealoffset->status_end = sidx;

            uint offset_size = (dimx+1)*(dimy+1) + 1;
            memcpy(gctx->h_offset+oidx, source->get_offset(), offset_size * sizeof(uint16_t));
            source->idealoffset->offset_start = oidx;
            oidx += offset_size;
            source->idealoffset->offset_end = oidx;

            uint edge_sequences_size = source->get_len_edge_sequences();
            memcpy(gctx->h_edge_sequences+eidx, source->get_edge_sequence(), edge_sequences_size * sizeof(EdgeSeq));
            source->idealoffset->edge_sequences_start = eidx;
            eidx += edge_sequences_size;
            source->idealoffset->edge_sequences_end = eidx;

            uint vertices_size = source->get_num_vertices();
            memcpy(gctx->h_vertices+vidx, source->get_boundary()->p, vertices_size * sizeof(Point));
            source->idealoffset->vertices_start = vidx;
            vidx += vertices_size;
            source->idealoffset->vertices_end = vidx;

            uint gridline_offset_size = source->get_vertical()->get_num_grid_lines();
            memcpy(gctx->h_gridline_offset+goidx, source->get_vertical()->get_offset(), gridline_offset_size * sizeof(uint16_t));
            source->idealoffset->gridline_offset_start = goidx;
            goidx += gridline_offset_size;
            source->idealoffset->gridline_offset_end = goidx;

            uint gridline_nodes_size = source->get_vertical()->get_num_crosses();
            memcpy(gctx->h_gridline_nodes+gnidx, source->get_vertical()->get_intersection_nodes(), gridline_nodes_size * sizeof(double));
            source->idealoffset->gridline_nodes_start = gnidx;
            gnidx += gridline_nodes_size;
            source->idealoffset->gridline_nodes_end = gnidx;
        }

    }

    for(auto &tp : gctx->polygon_pairs){
        flag2 = true;
        Ideal *source = tp.first;
        int dimx = source->get_dimx(), dimy = source->get_dimy();
        if(source->idealoffset == nullptr){
            source->idealoffset = new IdealOffset{};

            uint info_size = gctx->polygon_pairs.size();
            Idealinfo idealinfo{source->getMBB(), dimx, dimy, source->get_step_x(), source->get_step_y()};
            memcpy(gctx->h_info+iidx, &idealinfo, sizeof(Idealinfo));
            source->idealoffset->info_start = iidx;
            iidx ++;
            source->idealoffset->info_end = iidx;

            uint status_size = (dimx+1)*(dimy+1) / 4 + 1;
            memcpy(gctx->h_status+sidx, source->get_status(), status_size);
            source->idealoffset->status_start = sidx;
            sidx += status_size;
            source->idealoffset->status_end = sidx;

            uint offset_size = (dimx+1)*(dimy+1) + 1;
            memcpy(gctx->h_offset+oidx, source->get_offset(), offset_size * sizeof(uint16_t));
            source->idealoffset->offset_start = oidx;
            oidx += offset_size;
            source->idealoffset->offset_end = oidx;

            uint edge_sequences_size = source->get_len_edge_sequences();
            memcpy(gctx->h_edge_sequences+eidx, source->get_edge_sequence(), edge_sequences_size * sizeof(EdgeSeq));
            source->idealoffset->edge_sequences_start = eidx;
            eidx += edge_sequences_size;
            source->idealoffset->edge_sequences_end = eidx;

            uint vertices_size = source->get_num_vertices();
            memcpy(gctx->h_vertices+vidx, source->get_boundary()->p, vertices_size * sizeof(Point));
            source->idealoffset->vertices_start = vidx;
            vidx += vertices_size;
            source->idealoffset->vertices_end = vidx;

            uint gridline_offset_size = source->get_vertical()->get_num_grid_lines();
            memcpy(gctx->h_gridline_offset+goidx, source->get_vertical()->get_offset(), gridline_offset_size * sizeof(uint16_t));
            source->idealoffset->gridline_offset_start = goidx;
            goidx += gridline_offset_size;
            source->idealoffset->gridline_offset_end = goidx;

            uint gridline_nodes_size = source->get_vertical()->get_num_crosses();
            memcpy(gctx->h_gridline_nodes+gnidx, source->get_vertical()->get_intersection_nodes(), gridline_nodes_size * sizeof(double));
            source->idealoffset->gridline_nodes_start = gnidx;
            gnidx += gridline_nodes_size;
            source->idealoffset->gridline_nodes_end = gnidx;
        }

        Ideal *target = tp.second;   
	    dimx = target->get_dimx(), dimy = target->get_dimy();
        if(target->idealoffset == nullptr){
            target->idealoffset = new IdealOffset{};

            uint info_size = gctx->polygon_pairs.size();
            Idealinfo idealinfo{target->getMBB(), dimx, dimy, target->get_step_x(), target->get_step_y()};
            memcpy(gctx->h_info+iidx, &idealinfo, sizeof(Idealinfo));
            target->idealoffset->info_start = iidx;
            iidx ++;
            target->idealoffset->info_end = iidx;

            uint status_size = (dimx+1)*(dimy+1) / 4 + 1;
            assert((status_size+sidx) < 1U * BUFFER_SIZE);
            memcpy(gctx->h_status+sidx, target->get_status(), status_size);
            target->idealoffset->status_start = sidx;
            sidx += status_size;
            target->idealoffset->status_end = sidx;

            uint offset_size = (dimx+1)*(dimy+1) + 1;
            memcpy(gctx->h_offset+oidx, target->get_offset(), offset_size * sizeof(uint16_t));
            target->idealoffset->offset_start = oidx;
            oidx += offset_size;
            target->idealoffset->offset_end = oidx;

            uint edge_sequences_size = target->get_len_edge_sequences();
            assert((edge_sequences_size+eidx)*sizeof(EdgeSeq) < 1U * BUFFER_SIZE);
            memcpy(gctx->h_edge_sequences+eidx, target->get_edge_sequence(), edge_sequences_size * sizeof(EdgeSeq));
            target->idealoffset->edge_sequences_start = eidx;
            eidx += edge_sequences_size;
            target->idealoffset->edge_sequences_end = eidx;

            uint vertices_size = target->get_num_vertices();
            assert((vertices_size+vidx)*sizeof(Point) < (4ULL * BUFFER_SIZE));
            memcpy(gctx->h_vertices+vidx, target->get_boundary()->p, vertices_size * sizeof(Point));
            target->idealoffset->vertices_start = vidx;
            vidx += vertices_size;
            target->idealoffset->vertices_end = vidx;

            uint gridline_offset_size = target->get_vertical()->get_num_grid_lines();
            memcpy(gctx->h_gridline_offset+goidx, target->get_vertical()->get_offset(), gridline_offset_size * sizeof(uint16_t));
            target->idealoffset->gridline_offset_start = goidx;
            goidx += gridline_offset_size;
            target->idealoffset->gridline_offset_end = goidx;

            uint gridline_nodes_size = target->get_vertical()->get_num_crosses();
            memcpy(gctx->h_gridline_nodes+gnidx, target->get_vertical()->get_intersection_nodes(), gridline_nodes_size * sizeof(double));
            target->idealoffset->gridline_nodes_start = gnidx;
            gnidx += gridline_nodes_size;
            target->idealoffset->gridline_nodes_end = gnidx;
        }
    }

    assert(flag1 ^ flag2);

    CUDA_SAFE_CALL(hipMemcpy(gctx->d_info, gctx->h_info, BUFFER_SIZE / 4UL * sizeof(uint8_t), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(gctx->d_status, gctx->h_status, BUFFER_SIZE / 4UL * sizeof(uint8_t), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(gctx->d_offset, gctx->h_offset, BUFFER_SIZE / 2UL * sizeof(uint8_t), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(gctx->d_edge_sequences, gctx->h_edge_sequences, BUFFER_SIZE * sizeof(uint8_t), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(gctx->d_vertices, gctx->h_vertices, 4UL * BUFFER_SIZE * sizeof(uint8_t), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(gctx->d_gridline_offset, gctx->h_gridline_offset, BUFFER_SIZE / 4UL * sizeof(uint8_t), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(gctx->d_gridline_nodes, gctx->h_gridline_nodes, BUFFER_SIZE / 4UL * sizeof(uint8_t), hipMemcpyHostToDevice));

}