#include "hip/hip_runtime.h"
#include "geometry.cuh"

#define WITHIN_DISTANCE 10

struct Batch{
	uint s_start = 0;
	uint t_start = 0;
	uint s_length = 0;
	uint t_length = 0;
	int pair_id = 0;
};

__global__ void kernel_init(pair<IdealOffset, IdealOffset> *d_pairs, Idealinfo *d_info, uint size, double *distance){
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	if(x < size){
		pair<IdealOffset, IdealOffset> &pair = d_pairs[x];
		IdealOffset &source = pair.first;
		IdealOffset &target = pair.second;
		box &s_mbr = d_info[source.info_start].mbr;
        box &t_mbr = d_info[target.info_start].mbr;

		distance[x] = gpu_max_distance(s_mbr, t_mbr);
	}
}

__global__ void kernel_1(pair<IdealOffset, IdealOffset> *d_pairs, Idealinfo *d_info, uint8_t *d_status, PixMapping* d_pixpolypairs, uint *buffer_size, uint size, int *step, bool *resultmap){
    const int pair_id = blockIdx.x * blockDim.x + threadIdx.x;
    if(pair_id < size){
		// if(resultmap[pair_id] != 0) return;
        pair<IdealOffset, IdealOffset> &pair = d_pairs[pair_id];
        IdealOffset &source = pair.first;
        IdealOffset &target = pair.second;

        box &s_mbr = d_info[source.info_start].mbr, &t_mbr = d_info[target.info_start].mbr;
        const double &s_step_x = d_info[source.info_start].step_x, &s_step_y = d_info[source.info_start].step_y;
		const int &s_dimx = d_info[source.info_start].dimx, &s_dimy = d_info[source.info_start].dimy;

        int lowx = gpu_get_offset_x(s_mbr.low[0], t_mbr.low[0], s_step_x, s_dimx);
        int lowy = gpu_get_offset_y(s_mbr.low[1], t_mbr.low[1], s_step_y, s_dimy);
        int highx = gpu_get_offset_x(s_mbr.low[0], t_mbr.high[0], s_step_x, s_dimx);
        int highy = gpu_get_offset_y(s_mbr.low[1], t_mbr.high[1], s_step_y, s_dimy);


        if(*step == 0){
            for(int x = lowx; x <= highx; x ++ ){
                for(int y = lowy; y <= highy; y ++ ){
                    int id = gpu_get_id(x, y, s_dimx);
                    assert((id / (s_dimx+1)) <= s_dimy);
                    // if(gpu_show_status(d_status, source.status_start, id) == BORDER){
                        int idx = atomicAdd(buffer_size, 1);
                        d_pixpolypairs[idx].pair_id = pair_id;
                        d_pixpolypairs[idx].pix_id =  id;                        
                    // }
                }
            }
        }else{
            int ymin = max(0, lowy - *step);
            int ymax = min(s_dimy, highy + *step);

            // left scan      
            if(lowx - *step >= 0){
                for(int y = ymin; y <= ymax; y ++){
                    int id = gpu_get_id(lowx-*step, y, s_dimx);
                    assert((id / (s_dimx+1)) <= s_dimy);
                    // if(gpu_show_status(d_status, source.status_start, id) == BORDER){
                        int idx = atomicAdd(buffer_size, 1);
                        d_pixpolypairs[idx].pair_id = pair_id;
                        d_pixpolypairs[idx].pix_id =  id;  
                    // }
                }
            }
            // right scan
            if(highx + *step <= s_dimx){
                for(int y = ymin; y <= ymax; y++){
                    int id = gpu_get_id(highx+*step, y, s_dimx);
                    assert((id / (s_dimx+1)) <= s_dimy);
                    // if(gpu_show_status(d_status, source.status_start, id) == BORDER){
                        int idx = atomicAdd(buffer_size, 1);
                        d_pixpolypairs[idx].pair_id = pair_id;
                        d_pixpolypairs[idx].pix_id =  id;  
                    // }
                }
            }
            // skip the first if there is left scan
            int xmin = max(0, lowx - *step + (lowx - *step >= 0));
            // skip the last if there is right scan
            int xmax = min(s_dimx, highx + *step - (highx + *step <= s_dimx));  

            // bottom scan
            if (lowy - *step >= 0) {
                for (int x = xmin; x <= xmax; x++) {
                    int id = gpu_get_id(x, lowy-*step, s_dimx);
                    assert((id / (s_dimx+1)) <= s_dimy);
                    // if(gpu_show_status(d_status, source.status_start, id) == BORDER){
                        int idx = atomicAdd(buffer_size, 1);
                        d_pixpolypairs[idx].pair_id = pair_id;
                        d_pixpolypairs[idx].pix_id = id;
                    // }
                }
            }   
            // top scan
            if (highy + *step <= s_dimy) {
                for (int x = xmin; x <= xmax; x++) {
                    int id = gpu_get_id(x, highy+*step, s_dimx);
                    assert((id / (s_dimx+1)) <= s_dimy);
                    // if(gpu_show_status(d_status, source.status_start, id) == BORDER){
                        int idx = atomicAdd(buffer_size, 1);
                        d_pixpolypairs[idx].pair_id = pair_id;
                        d_pixpolypairs[idx].pix_id = id;
                    // }
                }
            }
        }
    }    
}

__global__ void kernel_2_1(PixMapping *d_pixpolypairs, pair<IdealOffset, IdealOffset> *d_pairs, Idealinfo *d_info, uint8_t *d_status, int *step, uint *size, PixPair *d_pixpairs, uint *buffer_size, bool *resultmap){
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    if(x < *size){
		int pair_id = d_pixpolypairs[x].pair_id;
		int cur = d_pixpolypairs[x].pix_id;
        // if(resultmap[pair_id]) return;

		pair<IdealOffset, IdealOffset> &pair = d_pairs[pair_id];
        IdealOffset &source = pair.first;
        IdealOffset &target = pair.second;

        box &s_mbr = d_info[source.info_start].mbr, &t_mbr = d_info[target.info_start].mbr;
		const double &s_step_x = d_info[source.info_start].step_x, &s_step_y = d_info[source.info_start].step_y;
		const int &s_dimx = d_info[source.info_start].dimx, &s_dimy = d_info[source.info_start].dimy;
        const double &t_step_x = d_info[target.info_start].step_x, &t_step_y = d_info[target.info_start].step_y;
        const int &t_dimx = d_info[target.info_start].dimx, &t_dimy = d_info[target.info_start].dimy;

        // if((cur / (s_dimx+1)) > s_dimy){
        //     printf("cur = %d, dimx = %d, dimy = %d\n", cur, s_dimx, s_dimy);
        //    assert((cur / (s_dimx+1)) <= s_dimy);
        // }
        auto pix_box = gpu_get_pixel_box(gpu_get_x(cur, s_dimx), gpu_get_y(cur, s_dimx, s_dimy), s_mbr.low[0], s_mbr.low[1], s_step_x, s_step_y);

        int lowx = gpu_get_offset_x(t_mbr.low[0], pix_box.low[0], t_step_x, t_dimx);
        int lowy = gpu_get_offset_y(t_mbr.low[1], pix_box.low[1], t_step_y, t_dimy);
        int highx = gpu_get_offset_x(t_mbr.low[0], pix_box.high[0], t_step_x, t_dimx);
        int highy = gpu_get_offset_y(t_mbr.low[1], pix_box.high[1], t_step_y, t_dimy);

        if(*step == 0){
            for(int x = lowx; x <= highx; x ++ ){
                for(int y = lowy; y <= highy; y ++ ){
                    int id = gpu_get_id(x, y, t_dimx);
                    // if(gpu_show_status(d_status, source.status_start, id) == BORDER){
                        int idx = atomicAdd(buffer_size, 1);
                        d_pixpairs[idx] = {cur, id, pair_id};                        
                    // }
                }
            }
        }else{

            int ymin = max(0, lowy - *step);
            int ymax = min(t_dimy, highy + *step);

            // left scan      
            if(lowx - *step >= 0){
                for(int y = ymin; y <= ymax; y ++){
                    int id = gpu_get_id(lowx-*step, y, t_dimx);
                    if(gpu_show_status(d_status, target.status_start, id) == BORDER){
                        int idx = atomicAdd(buffer_size, 1);
                        d_pixpairs[idx] = {cur, id, pair_id};
                    }
                }
            }
            // right scan
            if(highx + *step <= t_dimx){
                for(int y = ymin; y <= ymax; y++){
                    int id = gpu_get_id(highx+*step, y, t_dimx);
                    if(gpu_show_status(d_status, target.status_start, id) == BORDER){
                        int idx = atomicAdd(buffer_size, 1);
                        d_pixpairs[idx] = {cur, id, pair_id};
                    }
                }
            }
            // skip the first if there is left scan
            int xmin = max(0, lowx - *step + (lowx - *step >= 0));
            // skip the last if there is right scan
            int xmax = min(t_dimx, highx + *step - (highx + *step <= t_dimx));  

            // bottom scan
            if (lowy - *step >= 0) {
                for (int x = xmin; x <= xmax; x++) {
                    int id = gpu_get_id(x, lowy-*step, t_dimx);
                    if(gpu_show_status(d_status, target.status_start, id) == BORDER){
                        int idx = atomicAdd(buffer_size, 1);
                        d_pixpairs[idx] = {cur, id, pair_id};
                    }
                }
            }   
            // top scan
            if (highy + *step <= t_dimy) {
                for (int x = xmin; x <= xmax; x++) {
                    int id = gpu_get_id(x, highy+*step, t_dimx);
                    if(gpu_show_status(d_status, target.status_start, id) == BORDER){
                        int idx = atomicAdd(buffer_size, 1);
                        d_pixpairs[idx] = {cur, id, pair_id};
                    }
                }
            }
        }
	}	
}

__global__ void kernel_2_2(PixPair *d_pixpairs, pair<IdealOffset, IdealOffset> *d_pairs, uint16_t *d_offset, EdgeSeq *d_edge_sequences, uint *size, Batch *batches, uint *batch_size, bool *resultmap){
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    if(x < *size){
        int pair_id = d_pixpairs[x].pair_id;
        int p = d_pixpairs[x].source_pixid;
        int p2 = d_pixpairs[x].target_pixid;
        if(resultmap[pair_id]) return;

        pair<IdealOffset, IdealOffset> &pair = d_pairs[pair_id];
        IdealOffset &source = pair.first;
        IdealOffset &target = pair.second;

        int s_num_sequence = (d_offset+source.offset_start)[p + 1] - (d_offset+source.offset_start)[p];
        int t_num_sequence = (d_offset+target.offset_start)[p2 + 1] - (d_offset+target.offset_start)[p2];

        for(int i = 0; i < s_num_sequence; ++ i ){
            EdgeSeq r = (d_edge_sequences+source.edge_sequences_start)[(d_offset+source.offset_start)[p] + i];
            for(int j = 0; j < t_num_sequence; ++ j ){
                EdgeSeq r2 = (d_edge_sequences+target.edge_sequences_start)[(d_offset+target.offset_start)[p2] + j];
                if(r.length < 2 || r2.length < 2) continue;
                int max_size = 32;
                for(uint s = 0; s < r.length; s += max_size){
                    uint end_s = min(s + max_size, r.length);
                    for(uint t = 0; t < r2.length; t += max_size){
                        uint end_t = min(t + max_size, r2.length);
                        uint idx = atomicAdd(batch_size, 1U);
                        batches[idx].s_start = source.vertices_start+r.start + s;
                        batches[idx].t_start = target.vertices_start+r2.start + t;
                        batches[idx].s_length = end_s - s;
                        batches[idx].t_length = end_t - t;
                        batches[idx].pair_id = pair_id;
                    }
                }
            }
        }
    }
}

__global__ void kernel_2_3(Batch* batches, Point *d_vertices, uint *size, double *distance, bool* resultmap){
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	if(x < *size){
		uint s1  = batches[x].s_start;
		uint s2 = batches[x].t_start;
		uint len1 = batches[x].s_length;
		uint len2 = batches[x].t_length;
		int pair_id = batches[x].pair_id;
		if(resultmap[pair_id] != 0) return;

        double dist = gpu_segment_to_segment_within_batch(d_vertices+s1, d_vertices+s1, len1, len2);

        atomicMinDouble(distance+pair_id, dist);

        if(distance[pair_id] <= WITHIN_DISTANCE){
            resultmap[pair_id] = true;
            return;
        }


	}   
}

__global__ void kernel_3(pair<IdealOffset, IdealOffset> *d_pairs, Idealinfo *d_info, int *step, uint size, double *distance, bool *resultmap){
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	if(x < size){
        if(resultmap[x]) return;

		pair<IdealOffset, IdealOffset> &pair = d_pairs[x];
        IdealOffset &source = pair.first;
        IdealOffset &target = pair.second;

        box &s_mbr = d_info[source.info_start].mbr, &t_mbr = d_info[target.info_start].mbr;
		const double &s_step_x = d_info[source.info_start].step_x, &s_step_y = d_info[source.info_start].step_y;
		const int &s_dimx = d_info[source.info_start].dimx, &s_dimy = d_info[source.info_start].dimy;

        // min_possible = mbrdist + step * step_size
        double min_possible = gpu_distance(s_mbr, t_mbr) + (*step) * gpu_get_step(s_mbr, s_dimx, s_dimy);
        if(distance[x] <= min_possible){
            resultmap[x] = true;
            return;
        }



    }
}

uint cuda_within_polygon(query_context *gctx){
    CudaTimer timer;

    float sum_filter = 0.0;
	float sum_unroll = 0.0;
	float sum_refinement = 0.0;
	float sum_check = 0.0;

    uint size = gctx->polygon_pairs.size();

    pair<IdealOffset, IdealOffset> *h_pairs = new pair<IdealOffset, IdealOffset>[size];
    pair<IdealOffset, IdealOffset> *d_pairs = nullptr;

    for(int i = 0; i < size; i ++ ){
        Ideal *source = gctx->polygon_pairs[i].first;
        Ideal *target = gctx->polygon_pairs[i].second;
        h_pairs[i] = {*source->idealoffset, *target->idealoffset};
    }

    CUDA_SAFE_CALL(hipMalloc((void**) &d_pairs, size * sizeof(pair<IdealOffset, IdealOffset>)));
	CUDA_SAFE_CALL(hipMemcpy(d_pairs, h_pairs, size * sizeof(pair<IdealOffset, IdealOffset>), hipMemcpyHostToDevice));
    
	double *h_distance = new double[size * sizeof(double)];
    double *d_distance = nullptr;
    CUDA_SAFE_CALL(hipMalloc((void **) &d_distance, size * sizeof(double)));

	bool *d_resultmap = nullptr;
    CUDA_SAFE_CALL(hipMalloc((void **) &d_resultmap, size * sizeof(bool)));
	CUDA_SAFE_CALL(hipMemset(d_resultmap, 0, size * sizeof(bool)));

    int h_step = 0;
	int *d_step = nullptr;
	CUDA_SAFE_CALL(hipMalloc((void **) &d_step, sizeof(int)));
	CUDA_SAFE_CALL(hipMemset(d_step, 0, sizeof(int)));

    int h_step_inner = 0;
    int *d_step_inner = nullptr;
    CUDA_SAFE_CALL(hipMalloc((void **) &d_step_inner, sizeof(int)));
	CUDA_SAFE_CALL(hipMemset(d_step_inner, 0, sizeof(int)));

    PixMapping *d_pair_pixpoly = nullptr;
    CUDA_SAFE_CALL(hipMalloc((void **) &d_pair_pixpoly, 16*1024*1024*sizeof(PixMapping)));
    uint *d_pixpoly_size = nullptr;
    CUDA_SAFE_CALL(hipMalloc((void **) &d_pixpoly_size, sizeof(uint)));
    CUDA_SAFE_CALL(hipMemset(d_pixpoly_size, 0, sizeof(uint)));
    uint h_pixpoly_size;

    char *d_BufferInput = nullptr;
    CUDA_SAFE_CALL(hipMalloc((void **) &d_BufferInput, 4UL * 1024 * 1024 * 1024));
    uint *d_bufferinput_size = nullptr;
    CUDA_SAFE_CALL(hipMalloc((void **) &d_bufferinput_size, sizeof(uint)));
    CUDA_SAFE_CALL(hipMemset(d_bufferinput_size, 0, sizeof(uint)));
    uint h_bufferinput_size;

    char *d_BufferOutput = nullptr;
    CUDA_SAFE_CALL(hipMalloc((void **) &d_BufferOutput, 4UL * 1024 * 1024 * 1024));
    uint *d_bufferoutput_size = nullptr;
    CUDA_SAFE_CALL(hipMalloc((void **) &d_bufferoutput_size, sizeof(uint)));
    CUDA_SAFE_CALL(hipMemset(d_bufferoutput_size, 0, sizeof(uint)));
    uint h_bufferoutput_size;

    int grid_size_x = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
	dim3 block_size(BLOCK_SIZE, 1, 1);
	dim3 grid_size(grid_size_x, 1, 1);

    timer.startTimer();

	kernel_init<<<grid_size, block_size>>>(d_pairs, gctx->d_info, size, d_distance);
	hipDeviceSynchronize();
	check_execution("kernel init");

    timer.stopTimer();
    printf("kernel initialization time: %f ms\n", timer.getElapsedTime());

    printf("SIZE = %u\n", size);

    while(true){
        // h_step = 2;
        CUDA_SAFE_CALL(hipMemcpy(d_step, &h_step, sizeof(int), hipMemcpyHostToDevice));

        printf("STEP: %d\n", h_step);
		CUDA_SAFE_CALL(hipMemset(d_pixpoly_size, 0, sizeof(uint)));

        // timer.startTimer();

		kernel_1<<<grid_size, block_size>>>(d_pairs, gctx->d_info, gctx->d_status, d_pair_pixpoly, d_pixpoly_size, size, d_step, d_resultmap);
		hipDeviceSynchronize();
		check_execution("Kernel filter");

        // timer.stopTimer();
    	// printf("kernel_1 time: %f ms\n", timer.getElapsedTime());
		// sum_filter += timer.getElapsedTime();

		CUDA_SAFE_CALL(hipMemcpy(&h_pixpoly_size, d_pixpoly_size, sizeof(uint), hipMemcpyDeviceToHost));

        if(h_pixpoly_size == 0) break;	
        // printf("h_pixpoly_size: %u\n", h_pixpoly_size);
        h_step_inner = 0;
        while (true) {
            CUDA_SAFE_CALL(hipMemset(d_bufferinput_size, 0, sizeof(uint)));
            CUDA_SAFE_CALL(hipMemset(d_bufferoutput_size, 0, sizeof(uint)));

            grid_size.x = (h_pixpoly_size + BLOCK_SIZE - 1) / BLOCK_SIZE;

            kernel_2_1<<<grid_size, block_size>>>(d_pair_pixpoly, d_pairs, gctx->d_info, gctx->d_status, d_step_inner, d_pixpoly_size, (PixPair *)d_BufferInput, d_bufferinput_size, d_resultmap);
            hipDeviceSynchronize();
            check_execution("Kernel_2_1");

            CUDA_SAFE_CALL(hipMemcpy(&h_bufferinput_size, d_bufferinput_size, sizeof(uint), hipMemcpyDeviceToHost)); 
            printf("step = %d, h_bufferinput_size: %u\n", h_step_inner, h_bufferinput_size);
            if(h_bufferinput_size == 0) break;

    //         grid_size.x = (h_bufferinput_size + BLOCK_SIZE - 1) /
    //         BLOCK_SIZE;

    //         kernel_2_2<<<grid_size,
    //         block_size>>>((PixPair*)d_BufferInput, d_pairs,
    //         gctx->d_offset, gctx->d_edge_sequences,
    //         d_bufferinput_size, (Batch *)d_BufferOutput,
    //         d_bufferoutput_size, d_resultmap);
    //         hipDeviceSynchronize();
    // 	    check_execution("Kernel_2_2");

    //         swap(d_BufferInput, d_BufferOutput);
    //         swap(d_bufferinput_size, d_bufferoutput_size);
    //         CUDA_SAFE_CALL(hipMemset(d_bufferoutput_size, 0,
    //         sizeof(uint)));

    //         grid_size.x = (h_bufferinput_size + BLOCK_SIZE - 1) /
    //         BLOCK_SIZE;

    //         kernel_2_3<<<grid_size, block_size>>>((Batch
    //         *)d_BufferInput, gctx->d_vertices, d_bufferinput_size,
    //         d_distance, d_resultmap); hipDeviceSynchronize();
    // 	    check_execution("Kernel_2_3");

    //         h_step_inner ++;
    //         CUDA_SAFE_CALL(hipMemcpy(d_step_inner, &h_step_inner,
    //         sizeof(int), hipMemcpyHostToDevice));
        }

    //     grid_size.x = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    //     kernel_3<<<grid_size, block_size>>>(d_pairs, gctx->d_info, d_step, size, d_distance, d_resultmap);
    //     hipDeviceSynchronize();
    //     check_execution("Kernel_3");

        if(h_step == 2) break;
        h_step ++;
        CUDA_SAFE_CALL(hipMemcpy(d_step, &h_step, sizeof(int), hipMemcpyHostToDevice));
    }

    CUDA_SAFE_CALL(hipMemcpy(h_distance, d_distance, size * sizeof(double), hipMemcpyDeviceToHost));
	int found = 0;
	for(int i = 0 ;i < size; i ++){
		if(h_distance[i] <= WITHIN_DISTANCE) found ++;
	}

    return found;

}