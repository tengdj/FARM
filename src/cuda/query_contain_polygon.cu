#include "hip/hip_runtime.h"
#include "geometry.cuh"
#include "Ideal.h"

struct Task
{
    uint s_start = 0;
    uint t_start = 0;
    uint s_length = 0;
    uint t_length = 0;
    int pair_id = 0;
};

// flags: 0(not contain), 1(maybe contain), 2(contain)
__global__ void kernel_filter_contain_polygon(pair<uint32_t,uint32_t>* pairs, IdealOffset *idealoffset,
                                             RasterInfo *info, uint8_t *status, uint size, 
                                             PixPair *pixpairs, uint *pp_size, uint8_t *flags)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x >= size || flags[x] != 1) return;  

	const pair<uint32_t, uint32_t> pair = pairs[x];
    const uint32_t src_idx = pair.first;
	const uint32_t tar_idx = pair.second;
	const IdealOffset source = idealoffset[src_idx];
	const IdealOffset target = idealoffset[tar_idx];

	const box s_mbr = info[src_idx].mbr, t_mbr = info[tar_idx].mbr;				
	const double s_step_x = info[src_idx].step_x, s_step_y = info[src_idx].step_y; 
	const int s_dimx = info[src_idx].dimx, s_dimy = info[src_idx].dimy;			 
	const double t_step_x = info[tar_idx].step_x, t_step_y = info[tar_idx].step_y; 
	const int t_dimx = info[tar_idx].dimx, t_dimy = info[tar_idx].dimy;			

	uint itn = 0, etn = 0;	 
	bool flag_out = false; 

	int i_min = gpu_get_offset_x(s_mbr.low[0], t_mbr.low[0], s_step_x, s_dimx);
	int i_max = gpu_get_offset_x(s_mbr.low[0], t_mbr.high[0], s_step_x, s_dimx);
	int j_min = gpu_get_offset_y(s_mbr.low[1], t_mbr.low[1], s_step_y, s_dimy);
	int j_max = gpu_get_offset_y(s_mbr.low[1], t_mbr.high[1], s_step_y, s_dimy);

	for (int i = i_min; i <= i_max; i++)
	{
		for (int j = j_min; j <= j_max; j++)
		{
			int p = gpu_get_id(i, j, s_dimx);
			uint8_t source_status = gpu_show_status(status, source.status_start, p);

			itn += (source_status == IN);
            etn += (source_status == OUT);

			box bx = gpu_get_pixel_box(i, j, s_mbr.low[0], s_mbr.low[1], s_step_x, s_step_y);

			int _i_min = gpu_get_offset_x(t_mbr.low[0], bx.low[0], t_step_x, t_dimx);
			int _i_max = gpu_get_offset_x(t_mbr.low[0], bx.high[0], t_step_x, t_dimx);
			int _j_min = gpu_get_offset_y(t_mbr.low[1], bx.low[1], t_step_y, t_dimy);
			int _j_max = gpu_get_offset_y(t_mbr.low[1], bx.high[1], t_step_y, t_dimy);

			for (int _i = _i_min; _i <= _i_max; _i++)
			{
				for (int _j = _j_min; _j <= _j_max; _j++)
				{
					int p2 = gpu_get_id(_i, _j, t_dimx);
					uint8_t target_status = gpu_show_status(status, target.status_start, p2);

                    flag_out = ((source_status == OUT) && (target_status == IN));

					if (source_status == BORDER && target_status == BORDER)
					{
						int idx = atomicAdd(pp_size, 1U);
						pixpairs[idx].source_pixid = p;
						pixpairs[idx].target_pixid = p2;
						pixpairs[idx].pair_id = x;
					}
				}
			}
		}
	}

	uint total_pixels = (i_max - i_min + 1) * (j_max - j_min + 1);

	bool is_contained = (itn == total_pixels); 
	bool is_outside = (etn == total_pixels); 
    
    if(is_outside || flag_out) flags[x] = 0;
    return;
}

__global__ void kernel_unroll_contain_polygon(PixPair *pixpairs, pair<uint32_t, uint32_t> *pairs,
											 IdealOffset *idealoffset, uint8_t *status,
											 uint32_t *es_offset, EdgeSeq *edge_sequences,
											 uint *size, Task *tasks, uint *task_size)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx >= *size) return;
	
	int p = pixpairs[idx].source_pixid;
	int p2 = pixpairs[idx].target_pixid;
	int pair_id = pixpairs[idx].pair_id;

	const pair<uint32_t, uint32_t> pair = pairs[pair_id];
    const uint32_t src_idx = pair.first;
    const uint32_t tar_idx = pair.second;
	const IdealOffset source = idealoffset[src_idx];
    const IdealOffset target = idealoffset[tar_idx];

	uint s_offset_start = source.offset_start;
	uint t_offset_start = target.offset_start;
	uint s_edge_sequences_start = source.edge_sequences_start;
	uint t_edge_sequences_start = target.edge_sequences_start;

	int s_num_sequence = (es_offset + s_offset_start)[p + 1] - (es_offset + s_offset_start)[p];
	int t_num_sequence = (es_offset + t_offset_start)[p2 + 1] - (es_offset + t_offset_start)[p2];
	uint s_vertices_start = source.vertices_start;
	uint t_vertices_start = target.vertices_start;

	const int max_size = 16;

	for (int i = 0; i < s_num_sequence; ++i)
	{
		EdgeSeq r = (edge_sequences + s_edge_sequences_start)[(es_offset + s_offset_start)[p] + i];
		for (int j = 0; j < t_num_sequence; ++j)
		{
	 		EdgeSeq r2 = (edge_sequences + t_edge_sequences_start)[(es_offset + t_offset_start)[p2] + j];
			for (uint s = 0; s < r.length; s += max_size)
			{
				uint end_s = min(s + max_size, r.length);
				for (uint t = 0; t < r2.length; t += max_size)
				{
					uint end_t = min(t + max_size, r2.length);

					uint idx_task = atomicAdd(task_size, 1U);
					tasks[idx_task].s_start = s_vertices_start + r.start + s;
					tasks[idx_task].t_start = t_vertices_start + r2.start + t;
					tasks[idx_task].s_length = end_s - s;
					tasks[idx_task].t_length = end_t - t;
					tasks[idx_task].pair_id = pair_id;
	 			}
	 		}
		}
	}
}

__global__ void kernel_refinement_contain_polygon(Task *tasks, Point *d_vertices, uint *size, uint8_t *flags, uint *result)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x >= *size) return;
	
	uint s1 = tasks[x].s_start;
	uint s2 = tasks[x].t_start;
	uint len1 = tasks[x].s_length;
	uint len2 = tasks[x].t_length;
	int pair_id = tasks[x].pair_id;

	bool should_process = (flags[pair_id] == 1);

	bool has_intersection = should_process && gpu_segment_intersect_batch((d_vertices + s1), (d_vertices + s2), len1, len2);

	if (has_intersection) flags[pair_id] = 0;
}

__global__ void statistic_result(uint8_t *flags, uint size, uint *result){
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= size) return;
	if(flags[x] == 1) atomicAdd(result, 1);
}

void cuda_contain_polygon(query_context *gctx)
{
    uint h_bufferinput_size, h_bufferoutput_size;
	CUDA_SAFE_CALL(hipMemset(gctx->d_bufferinput_size, 0, sizeof(uint)));

	/*1. Raster Model Filtering*/
	int grid_size_x = (gctx->num_pairs + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 block_size(BLOCK_SIZE, 1, 1);
    dim3 grid_size(grid_size_x, 1, 1);

    kernel_filter_contain_polygon<<<grid_size, block_size>>>(gctx->d_candidate_pairs, gctx->d_idealoffset, gctx->d_info, gctx->d_status, gctx->num_pairs, (PixPair *)gctx->d_BufferInput, gctx->d_bufferinput_size, gctx->d_flags);
    hipDeviceSynchronize();
    check_execution("kernel_filter_contain_polygon");

    CUDA_SAFE_CALL(hipMemcpy(&h_bufferinput_size, gctx->d_bufferinput_size, sizeof(uint), hipMemcpyDeviceToHost));
    printf("h_buffer_size = %u\n", h_bufferinput_size);
	
    /*2. Unroll Refinement*/

    grid_size_x = (h_bufferinput_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    grid_size.x = grid_size_x;

    kernel_unroll_contain_polygon<<<grid_size, block_size>>>((PixPair *)gctx->d_BufferInput, gctx->d_candidate_pairs, gctx->d_idealoffset, gctx->d_status, gctx->d_offset, gctx->d_edge_sequences, gctx->d_bufferinput_size, (Task *)gctx->d_BufferOutput, gctx->d_bufferoutput_size);
    hipDeviceSynchronize();
    check_execution("kernel_unroll_contain_polygon");

    CUDA_SAFE_CALL(hipMemcpy(&h_bufferoutput_size, gctx->d_bufferoutput_size, sizeof(uint), hipMemcpyDeviceToHost));
    printf("h_buffer_size = %u\n", h_bufferoutput_size);
    
    /*3. Refinement step*/

    grid_size_x = (h_bufferoutput_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    grid_size.x = grid_size_x;

    kernel_refinement_contain_polygon<<<grid_size, block_size>>>((Task *)gctx->d_BufferOutput, gctx->d_vertices, gctx->d_bufferoutput_size, gctx->d_flags, gctx->d_result);
    hipDeviceSynchronize();
    check_execution("kernel_refinement_contain_polygon");

    grid_size_x = (gctx->num_pairs + BLOCK_SIZE - 1) / BLOCK_SIZE;
    grid_size.x = grid_size_x;

    statistic_result<<<grid_size, block_size>>>(gctx->d_flags, gctx->num_pairs, gctx->d_result);
    hipDeviceSynchronize();
    check_execution("statistic_result");

	// uint8_t* h_Buffer = new uint8_t[gctx->num_pairs];
    // CUDA_SAFE_CALL(hipMemcpy(h_Buffer, gctx->d_flags, gctx->num_pairs * sizeof(uint8_t), hipMemcpyDeviceToHost));
	// int _sum = 0;
    // for (int i = 0; i < gctx->num_pairs; i++) {
	// 	if(h_Buffer[i] == 2) _sum ++;
	// 	std::cout << (int)h_Buffer[i] << " ";
	// 	if ((i + 1) % 5 == 0) printf("\n");
    // }
    // printf("\n");

	// printf("sum = %d\n", _sum);

	CUDA_SAFE_CALL(hipMemcpy(&gctx->found, gctx->d_result, sizeof(uint), hipMemcpyDeviceToHost));

    return;
}
