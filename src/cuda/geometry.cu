#include "hip/hip_runtime.h"

#include "cuda_util.h"
#include "Ideal.h"
#include "mygpu.h"

#define BUFFER_SIZE 1024 * 1024 * 1024

void cuda_create_buffer(query_context *gctx, gpu_info *gpu){
	size_t size = BUFFER_SIZE;
	log("CPU momory:");
	gctx->h_status = new uint8_t[size];
	log("\t%.2f MB\tstatus buffer",1.0*size/1024/1024);

	log("GPU memory:");
	gpu->clear();
	gctx->d_status = (uint8_t *)gpu->allocate(size);
	log("\t%.2f MB\tstatus buffer",1.0*size/1024/1024);
}

void cuda_transfer_data(query_context *gctx){
	在完成MBR过滤以后才需要把需要用到的status传进buffer
}

__device__
bool check_contain(const double *polygon1, const double *polygon2, int num_vertices_1, int num_vertices_2){
	bool val = false;
	for(int p = 0;p<num_vertices_2-1;p++){
		double px = polygon2[p];
		double py = polygon2[num_vertices_2+p];
		for (int i = 0, j = 1; i < num_vertices_1-1; i++,j++) {
			// segment i->j intersect with line y=p.y
			double pyi = polygon1[num_vertices_1+i];
			double pyj = polygon1[num_vertices_1+j];
			if ((pyj>py) != (pyi>py))
			{
				double pxi = polygon1[i];
				double pxj = polygon1[j];
				double a = (pxj-pxi) / (pyj-pyi);
				if(px-pxi<a*(py-pyi)){
					val = !val;
				}
			}
		}
	}
	return val;
}

__global__
void contain_cuda(const double *poly1, const double *poly2, const uint *offset_size, int *ret, size_t num_pairs){

	// which polygon-polygon pair
	int pair_id = blockIdx.x*blockDim.x+threadIdx.x;
	if(pair_id>=num_pairs){
		return;
	}

	uint off1 = offset_size[pair_id*4];
	uint size1 = offset_size[pair_id*4+1];
	uint off2 = offset_size[pair_id*4+2];
	uint size2 = offset_size[pair_id*4+3];
	ret[pair_id] = (int)check_contain(poly1+off1,poly2+off2,size1,size2);
}

///*
// * data: contains the segments of the meshes mentioned in this join.
// * offset_size:  contains the offset in the data for each batch, and the sizes of two data sets
// * result: for the returned results for each batch
// * batch_num: number of computed batches
// *
// * */
//void contain_batch_gpu(gpu_info *gpu, double *data, uint *offset_size, int *result, size_t total_vertice_num, int pair_num){
//
//	assert(gpu);
//	hipSetDevice(gpu->device_id);
//	struct timeval start = get_cur_time();
//
//	// space for the results in GPU
//	int *d_ret = gpu->get_result(sizeof(int)*pair_num);
//	// space for the offset and size information in GPU
//	uint *d_os = gpu->get_os(sizeof(uint)*pair_num*4);
//	double *d_poly1 = gpu->source_data;
//	double *d_poly2 = gpu->get_data(total_vertice_num*2*sizeof(double));
//
//	CUDA_SAFE_CALL(hipMemcpy(d_poly2, data, total_vertice_num*2*sizeof(double), hipMemcpyHostToDevice));
//	CUDA_SAFE_CALL(hipMemcpy(d_os, offset_size, pair_num*4*sizeof(uint), hipMemcpyHostToDevice));
//	//logt("allocating data", start);
//
//	// compute the vectors of segments in data, save to d_vec
//	contain_cuda<<<pair_num/1024+1,1024>>>(d_poly1, d_poly2, d_os, d_ret, pair_num);
//	check_execution();
//	hipDeviceSynchronize();
//	//logt("computations", start);
//	CUDA_SAFE_CALL(hipMemcpy(result, d_ret, pair_num*sizeof(int), hipMemcpyDeviceToHost));
//	//logt("copy data out", start);
//}

__device__
double cuda_degree_per_kilometer_longitude(double latitude, double *degree_per_kilometer){
	double absla = abs(latitude);
	assert(absla<=90);
	if(absla==90){
		absla = 89.9;
	}
	return degree_per_kilometer[(int)(absla*10)];
}


__device__
double cuda_point_to_segment_distance(const Point &p, const Point &p1, const Point &p2, double *degree_per_kilometer) {

  double A = p.x - p1.x;
  double B = p.y - p1.y;
  double C = p2.x - p1.x;
  double D = p2.y - p1.y;

  double dot = A * C + B * D;
  double len_sq = C * C + D * D;
  double param = -1;
  if (len_sq != 0) //in case of 0 length line
      param = dot / len_sq;

  double xx, yy;

  if (param < 0) {
    xx = p1.x;
    yy = p1.y;
  } else if (param > 1) {
    xx = p2.x;
    yy = p2.y;
  } else {
    xx = p1.x + param * C;
    yy = p1.y + param * D;
  }

  double dx = p.x - xx;
  double dy = p.y - yy;
  dx = dx/cuda_degree_per_kilometer_longitude(p.y, degree_per_kilometer);
  dy = dy/degree_per_kilometer_latitude;

  return sqrt(dx * dx + dy * dy);
}

__device__
void atomicMin_double(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*) address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
            __double_as_longlong(fmin(val, __longlong_as_double(assumed))));
    } while (assumed != old);
}

__global__
void cuda_distance(double *dist, const Point p, const Point *vs, size_t vs_length,double *degree_per_kilometer){

	// which polygon-polygon pair
	int pair_id = blockIdx.x*blockDim.x+threadIdx.x;
	if(pair_id>=vs_length){
		return;
	}
	double d = cuda_point_to_segment_distance(p, vs[pair_id], vs[pair_id+1],degree_per_kilometer);
	//if(d>0.00001)
	{
		atomicMin_double(dist, d);
	}
}

// double point_to_segment_sequence_distance_gpu(Point &p, Point *vs, size_t seq_len, bool geography){
// 	assert(gpus.size()>0);
// 	gpu_info *gpu = gpus[0];
// 	Point *vs_d = (Point *)gpu->allocate(seq_len* sizeof(Point));
// 	double *dist_d = (double *)gpu->allocate(sizeof(double));
// 	double dist = DBL_MAX;
// 	CUDA_SAFE_CALL(hipMemcpy((char *)dist_d, (char *)&dist, sizeof(double),hipMemcpyHostToDevice));
// 	CUDA_SAFE_CALL(hipMemcpy((char *)vs_d, (char *)vs, seq_len*sizeof(Point),hipMemcpyHostToDevice));
// 	cuda_distance<<<seq_len/1024+1, 1024>>>(dist_d, p, vs_d, seq_len,gpu->degree_per_kilometer);
// 	CUDA_SAFE_CALL(hipMemcpy(&dist, dist_d, sizeof(double), hipMemcpyDeviceToHost));
// 	gpu->free((char *)dist_d, sizeof(double));
// 	gpu->free((char *)vs_d, seq_len* sizeof(Point));
// 	return dist;
// }



//void load_source_togpu(gpu_info *gpu, vector<MyPolygon *> &source){
//	size_t source_size = 0;
//	for(MyPolygon *p:source){
//		source_size += 2*p->boundary->num_vertices*sizeof(double);
//	}
//	gpu->get_source(source_size);
//	source_size = 0;
//	for(MyPolygon *p:source){
//		int num_vertices = p->boundary->num_vertices;
//		p->offset = source_size/sizeof(double);
//		CUDA_SAFE_CALL(hipMemcpy((char *)(gpu->source_data)+source_size, (char *)(p->boundary->x), num_vertices*sizeof(double),hipMemcpyHostToDevice));
//		source_size += num_vertices*sizeof(double);
//		CUDA_SAFE_CALL(hipMemcpy((char *)(gpu->source_data)+source_size, (char *)(p->boundary->y), num_vertices*sizeof(double),hipMemcpyHostToDevice));
//		source_size += num_vertices*sizeof(double);
//	}
//}
////
////
////void contain_batch_gpu(gpu_info *gpu, double *data, uint *offset_size, int *result, size_t total_vertice_num, int pair_num);
//////void load_source_togpu(gpu_info *gpu, vector<MyPolygon *> &source);
////
//int process_with_gpu(gpu_info *gpu, query_context *ctx){
//
//	int pair_num = ctx->candidates.size();
//	if(pair_num==0){
//		return 0;
//	}
//	int *result = new int[pair_num];
//	uint *offset_size = new uint[pair_num*4];
//	uint dataoffset = 0;
//	uint total_num_vertices = 0;
//	for(int i=0;i<pair_num;i++){
//		if(i==0||ctx->candidates[i].second->getid()!=ctx->candidates[i-1].second->getid()){
//			total_num_vertices += ctx->candidates[i].second->boundary->num_vertices;
//		}
//	}
//
//	double *tmpdata = new double[total_num_vertices*2];
//	for(int i=0;i<pair_num;i++){
//		offset_size[i*4] = ctx->candidates[i].first->offset;
//		offset_size[i*4+1] = ctx->candidates[i].first->boundary->num_vertices;
//		offset_size[i*4+3] = ctx->candidates[i].second->boundary->num_vertices;
//		if(i==0||ctx->candidates[i].second->getid()!=ctx->candidates[i-1].second->getid()){
//			offset_size[i*4+2] = dataoffset;
//			int num_vertices = ctx->candidates[i].second->boundary->num_vertices;
//			memcpy((char *)(tmpdata+dataoffset), (char *)(ctx->candidates[i].second->boundary->x), num_vertices*sizeof(double));
//			dataoffset += num_vertices;
//			memcpy((char *)(tmpdata+dataoffset), (char *)(ctx->candidates[i].second->boundary->y), num_vertices*sizeof(double));
//			dataoffset += num_vertices;
//		}else{
//			offset_size[i*4+2] = dataoffset-offset_size[i*4+3]*2;
//		}
//	}
//	assert(dataoffset==total_num_vertices*2);
//	ctx->candidates.clear();
//
//	int found = 0;
//	pthread_mutex_lock(&gpu->lock);
//	contain_batch_gpu(gpu,tmpdata,offset_size,result,total_num_vertices,pair_num);
//	pthread_mutex_unlock(&gpu->lock);
//	for(int i=0;i<pair_num;i++){
//		found += result[i];
//	}
//	delete []result;
//	delete []offset_size;
//	delete []tmpdata;
//	return found;
//}

