#include "hip/hip_runtime.h"
#include "../include/Ideal.h"
#include "cuda_util.h"
#include <thrust/device_vector.h>
#include <rtspatial/spatial_index.cuh>

#include <optix_function_table_definition.h>

__global__ void PrintResults(pair<uint32_t, uint32_t>* results, uint size){
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x < size){
        printf("%u %u\n", results[x].first, results[x].second);
    }
}

void indexFilter(query_context *gctx){
    int limit_box = std::numeric_limits<int>::max();
    int limit_query = std::numeric_limits<int>::max();
    vector<box> boxes;
    for (auto polygon : gctx->source_ideals)
    {
        boxes.push_back(*polygon->getMBB());
    }

	thrust::device_vector<rtspatial::Envelope<rtspatial::Point<coord_t, 2>>> d_boxes;
    CopyBoxes(boxes, d_boxes);

    rtspatial::SpatialIndex<coord_t, 2> index;
    rtspatial::Config config;
    rtspatial::Stream stream;
    rtspatial::Stopwatch sw;

    config.ptx_root = "/home/qmh/IDEAL/src/index/ptx";
    config.prefer_fast_build_query = false;
    config.max_geometries = d_boxes.size();

    index.Init(config);
    sw.start();
    index.Insert(
        rtspatial::ArrayView<rtspatial::Envelope<rtspatial::Point<coord_t, 2>>>(d_boxes),
        stream.hip_stream());
    stream.Sync();
    sw.stop();

    double t_load = sw.ms(), t_query;
    size_t n_results;
    rtspatial::Queue<thrust::pair<uint32_t, uint32_t>> results;
    rtspatial::SharedValue<rtspatial::Queue<thrust::pair<uint32_t, uint32_t>>::device_t> d_results;

    if(gctx->query_type == QueryType::contain){
        thrust::device_vector<rtspatial::Point<coord_t, 2>> d_queries;
        results.Init(std::max(
            1ul, (size_t)(boxes.size() * gctx->target_num)));
        d_results.set(stream.hip_stream(), results.DeviceObject());

        CopyPoints(gctx->points, gctx->target_num, d_queries);
        std::cout << "Loaded point queries " << gctx->target_num << std::endl;

        sw.start();
        index.Query(rtspatial::Predicate::kContains, rtspatial::ArrayView<rtspatial::Point<coord_t, 2>>(d_queries),
                    d_results.data(), stream.hip_stream());
                
    }else if(gctx->query_type == QueryType::contain_polygon){
        vector<box> queries;
        for(auto polygon : gctx->target_ideals)
        {
            queries.push_back(*polygon->getMBB());
        }

        thrust::device_vector<rtspatial::Envelope<rtspatial::Point<coord_t, 2> > > d_queries;
        results.Init(std::max(
            1ul, (size_t) (boxes.size() * queries.size())));
        d_results.set(stream.hip_stream(), results.DeviceObject());
        
        CopyBoxes(queries, d_queries);
        std::cout << "Loaded box queries " << queries.size() << std::endl;
        
        rtspatial::ArrayView<rtspatial::Envelope<rtspatial::Point<coord_t, 2> > > v_queries(d_queries);
        
        sw.start();
        index.Query(rtspatial::Predicate::kContains, v_queries, d_results.data(),
                    stream.hip_stream());

    }else{
        vector<box> queries;
        for (auto polygon : gctx->target_ideals)
        {
            queries.push_back(polygon->getMBB()->expand(gctx->within_distance, true));
        }

        thrust::device_vector<rtspatial::Envelope<rtspatial::Point<coord_t, 2> > > d_queries;
        results.Init(std::max(
            1ul, (size_t) (boxes.size() * queries.size())));
        d_results.set(stream.hip_stream(), results.DeviceObject());

        CopyBoxes(queries, d_queries);
        std::cout << "Loaded box queries " << queries.size() << std::endl;

        rtspatial::ArrayView<rtspatial::Envelope<rtspatial::Point<coord_t, 2> > > v_queries(d_queries);
        
        sw.start();
        index.Query(rtspatial::Predicate::kIntersects, v_queries, d_results.data(),
                    stream.hip_stream());

    }

    n_results = results.size(stream.hip_stream());
    sw.stop();
    t_query = sw.ms();
    std::cout << "RT, load " << t_load << " ms, query " << t_query
              << " ms, results: " << n_results << std::endl;

    auto d_result_ptr = results.data();
    CUDA_SAFE_CALL(hipMalloc((void **)&gctx->d_candidate_pairs, n_results * sizeof(pair<uint32_t, uint32_t>)));
    gctx->num_pairs = n_results;
    hipMemcpy(gctx->d_candidate_pairs, d_result_ptr, n_results * sizeof(pair<uint32_t, uint32_t>), hipMemcpyDeviceToDevice);

    // int grid_size_x = (n_results + 256 - 1) / 256;
	// dim3 block_size(256, 1, 1);
	// dim3 grid_size(grid_size_x, 1, 1);
    // PrintResults<<<grid_size, block_size>>>(gctx->d_candidate_pairs, gctx->num_pairs);
    // hipDeviceSynchronize();

    // pair<uint32_t, uint32_t>* h_candidate_pairs = new pair<uint32_t, uint32_t>[n_results];
    // hipMemcpy(h_candidate_pairs, gctx->d_candidate_pairs, n_results * sizeof(pair<uint32_t, uint32_t>), hipMemcpyDeviceToHost);
    // for(int i = 0; i < n_results; i ++){
    //     printf("pair%d\n", i);
    //     int source = h_candidate_pairs[i].first;
    //     int target = h_candidate_pairs[i].second;
    //     gctx->source_ideals[source]->MyPolygon::print();
    //     gctx->target_ideals[target]->MyPolygon::print();
    // }
    
}